
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello from CUDA Kernel!\n");
	printf("Thread ID: %d\n", threadIdx.x);
	printf("Block ID: %d\n", blockIdx.x);
	printf("Block Dim: %d\n", blockDim.x);
	printf("Grid Dim: %d\n", gridDim.x);
}

int main() {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();  // Wait for the kernel to finish

    return 0;
}


// Compile with: nvcc -o hello_cuda hello_cuda.cu
// Run with: ./hello_cuda
// Output:
// Hello from CUDA Kernel!
// Thread ID: 0
// Block ID: 0
// Block Dim: 1
// Grid Dim: 1